
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

using std::cout;
using std::endl;

__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int N)
{
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

	float result = 0.0;
	for (int i = 0; i < N; ++i)
	{
		result += A[row * N + i] * B[i * N + col];
	}

	C[(row * N) + col] = result;
}

float *initializeMatrix(unsigned int size)
{
	float *matrix = new float[size * size];
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
	    	matrix[(i * size) + j] = rand() % 100;
	    }
	}
	return matrix;
}

float *allocateDeviceMemory(int bufferSize)
{
    float *device;
	hipMalloc(&device, bufferSize);
	return device;
}

void copyHostMemoryToDevice(float *host, float *device, int bufferSize)
{
	hipMemcpy(device, host, bufferSize, hipMemcpyHostToDevice);
}

void createTimerEvents(hipEvent_t &start, hipEvent_t &stop)
{
    hipEventCreate(&start);
    hipEventCreate(&stop);
}

void destroyTimerEvents(hipEvent_t &start, hipEvent_t &stop)
{
	hipEventDestroy(start);
    hipEventDestroy(stop);
}

void freeMemory(float *devA, float *hostA, float *devB, float *hostB, float *devC)
{
    hipFree(devA);
    free(hostA);

    hipFree(devB);
    free(hostB);

    hipFree(devC);
}

void startTimer(hipEvent_t &start)
{
    hipEventRecord(start, 0);
}

void stopTimer(hipEvent_t &stop)
{
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
}

float readExecutionTimeInMillis(hipEvent_t &start, hipEvent_t &stop)
{
    float time;
    hipEventElapsedTime(&time, start, stop);
    return time;
}

int main(int argc, char *argv[])
{
	if (argc != 3)
	{
		std::cerr << "Usage: ./macierz_cuda threadCount matrixSize" << endl;
		return -1;
	}

	unsigned int threadCount = atoi(argv[1]);
	unsigned int matrixSize = atoi(argv[2]);

    float *hostA = initializeMatrix(matrixSize);
    float *hostB = initializeMatrix(matrixSize);

	int allocBuffer = matrixSize * matrixSize * sizeof(float);
	float *devA = allocateDeviceMemory(allocBuffer);
	float *devB = allocateDeviceMemory(allocBuffer);
	float *devC = allocateDeviceMemory(allocBuffer);

	copyHostMemoryToDevice(hostA, devA, allocBuffer);
	copyHostMemoryToDevice(hostB, devB, allocBuffer);

    hipEvent_t start, stop;
	createTimerEvents(start, stop);
	
	dim3 block(threadCount);
	dim3 grid(matrixSize / block.x, matrixSize / block.y);
	
    startTimer(start);

    matrixMultiplyKernel<<<grid, block>>>(devA, devB, devC, matrixSize);

    stopTimer(stop);
	
    cout << readExecutionTimeInMillis(start, stop) << endl;

    destroyTimerEvents(start, stop);
    freeMemory(devA, hostA, devB, hostB, devC);
    
    return 0;
}

