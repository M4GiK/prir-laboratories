
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

using std::cout;
using std::endl;

__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int N)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * 1024 * 1024;
	int row = threadId / N;
	int col = threadId % N;

	float result = 0.f;

	for (int i = 0; i < N; ++i)
	{
        if (row < N && col < N)
        {
	        result += A[(row * N) + i] * B[(i * N) + col];
        }
	}

	C[(row * N) + col] = result;
}

float *initializeMatrix(unsigned int size)
{
	float *matrix = new float[size * size];
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
	    	matrix[(i * size) + j] = rand() % 100;
	    }
	}
	return matrix;
}

float *allocateDeviceMemory(int bufferSize)
{
    float *device;
	hipMalloc(&device, bufferSize);
	return device;
}

void copyHostMemoryToDevice(float *host, float *device, int bufferSize)
{
	hipMemcpy(device, host, bufferSize, hipMemcpyHostToDevice);
}

void createTimerEvents(hipEvent_t &start, hipEvent_t &stop)
{
    hipEventCreate(&start);
    hipEventCreate(&stop);
}

void destroyTimerEvents(hipEvent_t &start, hipEvent_t &stop)
{
	hipEventDestroy(start);
    hipEventDestroy(stop);
}

void freeMemory(float *devA, float *hostA, float *devB, float *hostB, float *devC)
{
    hipFree(devA);
    free(hostA);

    hipFree(devB);
    free(hostB);

    hipFree(devC);
}

void startTimer(hipEvent_t &start)
{
    hipEventRecord(start, 0);
}

void stopTimer(hipEvent_t &stop)
{
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
}

float readExecutionTime(hipEvent_t &start, hipEvent_t &stop)
{
    float time;
    hipEventElapsedTime(&time, start, stop);
    return time;
}

int main(int argc, char *argv[])
{
	if (argc != 3)
	{
		std::cerr << "Usage: ./macierz_cuda threadCount matrixSize" << endl;
		return -1;
	}

	unsigned int matrixSize = atoi(argv[1]);
	unsigned int threadCount = atoi(argv[2]);

	dim3 dimBlock;
	dimBlock.x = matrixSize;
	dimBlock.y = matrixSize;

    float *hostA = initializeMatrix(matrixSize);
    float *hostB = initializeMatrix(matrixSize);

	int allocBuffer = matrixSize * matrixSize * sizeof(float);
	float *devA = allocateDeviceMemory(allocBuffer);
	float *devB = allocateDeviceMemory(allocBuffer);
	float *devC = allocateDeviceMemory(allocBuffer);

	copyHostMemoryToDevice(hostA, devA, allocBuffer);
	copyHostMemoryToDevice(hostB, devB, allocBuffer);

    hipEvent_t start, stop;
	createTimerEvents(start, stop);

    startTimer(start);

    matrixMultiplyKernel<<<dimBlock, threadCount>>>(devA, devB, devC, matrixSize);

    stopTimer(stop);
	
    cout << readExecutionTime(start, stop) << endl;

    destroyTimerEvents(start, stop);
    freeMemory(devA, hostA, devB, hostB, devC);
    
    return 0;
}

