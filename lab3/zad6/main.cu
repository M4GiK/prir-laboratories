#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	void *d = NULL;
	int i;
	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}

//============================================================================
// Name        : gauss_gpu
// Author      : Michał Szczygieł & Aleksander Śmierciak
//============================================================================
/*
#include <chrono>
#include <iostream>
#include <fstream>
#include <stdexcept>
#include <string>
#include <vector>

using std::cout;
using std::cin;
using std::endl;
using std::string;

typedef std::chrono::time_point<std::chrono::system_clock> TimePoint;
typedef std::chrono::duration<double> Duration;
typedef std::vector<double> DoubleMatrix;

void assertFileExist(const std::ios &input)
{
	if (!input.good())
	{
		throw new std::invalid_argument("Input file was not found.");
	}
}

void loadFile(string filePath)
{
	std::ifstream fileStream (filePath);
	assertFileExist (fileStream);
}

void performGaussianBlur(unsigned int threadCount, string videoInput,
		string videoOutput)
{
	loadFile(videoInput);

}

int main(int argc, char* argv[])
{
	if (argc != 4)
	{
		std::cerr << "Usage: ./gauss_gpu threadCount videoInput videoOutput"
				<< endl;
		return -1;
	}

	unsigned int threadCount = std::stoi(argv[1]);
	string videoInput = argv[2];
	string videoOutput = argv[3];

	performGaussianBlur(threadCount, videoInput, videoOutput);

	return 0;
}
*/
